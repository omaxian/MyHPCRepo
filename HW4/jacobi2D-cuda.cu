// 2D Poisson solver using Jacobi methods, CUDA version
// Ondrej Maxian

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <iostream>


double computeResidual(long N, double *u, double *f){
 double hsq = 1.0/((N+1.0)*(N+1.0));
 double res=0;
 //Timer t1;
 //t1.tic();
 #pragma omp parallel for default(none) \
 shared(u,f,hsq,N) \
 reduction(+:res)
 for (long j = 1; j <= N; j++) {
  for (long i = 1; i <= N; i++) {
	double resi=(-u[(N+2)*j+i-1]-u[(N+2)*(j-1)+i]
		-u[(N+2)*j+i+1]-u[(N+2)*(j+1)+i]+4*u[(N+2)*j+i])/hsq-f[(N+2)*j+i];
	res+=resi*resi;
  }
 }
 //std::cout << "Time to calc residual: " << t1.toc() << std::endl;
 res=sqrt(res);
 return res;
}
  

void Jacobi( long N, double *u, double *f, int maxiters) {
  double *ukp1=(double*) malloc((N+2)*(N+2) * sizeof(double));
  double hsq = 1.0/((N+1.0)*(N+1.0));
  double res=0;
  double relres=1e6;
  int iter=0;
  // Compute the initial residual
  double res0=computeResidual(N,u,f);
  std::cout << "Initial residual: " << res0 << std::endl;
  while (relres > 1e-6 && iter < maxiters){
   // Do the iteration
   //Timer t1;
   //t1.tic();
   # pragma omp parallel shared(ukp1,u,hsq,N,f) 
   { // start parallel region
   # pragma omp for
   for (long j = 1; j <= N; j++) {
    for (long i = 1; i <= N; i++) {
	 ukp1[(N+2)*j+i]=0.25*(hsq*f[(N+2)*j+i]+u[(N+2)*j+i-1]+u[(N+2)*(j-1)+i]
		+u[(N+2)*j+i+1]+u[(N+2)*(j+1)+i]);
    }
   }
   # pragma omp for
   for (long j = 1; j <= N; j++) {
    for (long i = 1; i <= N; i++) {
	 u[(N+2)*j+i]=ukp1[(N+2)*j+i];
    }
   }
   } // end parallel region
   //std::cout << "Time to do iteration and copy: " << t1.toc() << std::endl;
   res=computeResidual(N,u,f);
   iter++;
   relres=res/res0;
   if (iter%100==0)printf("CPU %10d %10f \n", iter, relres);
   }
  free(ukp1);
}

#define BLOCK_SIZE 1024

__global__ void gpJacobi(double* up1, const double* u, const double *f, long N){
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;
  int j = idx/(N+2);
  int i = idx % (N+2);
  double hsq = 1.0/((N+1.0)*(N+1.0));
  if (i >= 1 && i <= N && j >= 1 && j <= N){
  	up1[(N+2)*j+i]=0.25*(hsq*f[(N+2)*j+i]+u[(N+2)*j+i-1]+u[(N+2)*(j-1)+i]
		+u[(N+2)*j+i+1]+u[(N+2)*(j+1)+i]);
  }
}

__global__ void resvec(double* resv, const double* u, const double *f, long N){
  __shared__ double smem[BLOCK_SIZE];
  double hsq = 1.0/((N+1.0)*(N+1.0));
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;
  int j = idx/(N+2);
  int i = idx % (N+2);
  if (i >= 1 && i <= N && j >= 1 && j <= N){
	  double resi=(-u[(N+2)*j+i-1]-u[(N+2)*(j-1)+i]
		-u[(N+2)*j+i+1]-u[(N+2)*(j+1)+i]+4*u[(N+2)*j+i])/hsq-f[(N+2)*j+i];
	  smem[threadIdx.x]=resi*resi;
  } else{
	 smem[threadIdx.x]=0;
  }

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) resv[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void reduction_kernel2(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}


int main(int argc, char** argv) {
    int N;
    std::cout << "N? : " << std::endl;
    std::cin >> N;
    double *u, *f, *ugp;
    hipHostMalloc((void**)&u, (N+2)*(N+2) * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&f, (N+2)*(N+2) * sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&ugp, (N+2)*(N+2) * sizeof(double), hipHostMallocDefault);
    // Initialize u and f 
    #pragma omp parallel for schedule(static)
    for (int i = 0; i < (N+2)*(N+2); i++){ 
	u[i]=0.0;
	f[i]=1.0;
        ugp[i]=0.0;
    }
    int maxiters=1000;
    double tt = omp_get_wtime();
    // CPU Version
    Jacobi(N,u,f,maxiters);
    printf("CPU time = %f s\n", (omp_get_wtime()-tt));
    // GPU Version
    double *u_d, *up1_d, *f_d, *sum_d;
    hipMalloc(&u_d, (N+2)*(N+2)*sizeof(double));
    hipMalloc(&up1_d, (N+2)*(N+2)*sizeof(double));
    hipMalloc(&f_d, (N+2)*(N+2)*sizeof(double));
    hipMalloc(&sum_d, (N+2)*(N+2)*sizeof(double));
    hipMemcpyAsync(u_d, ugp, (N+2)*(N+2)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(up1_d, ugp, (N+2)*(N+2)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(f_d, f, (N+2)*(N+2)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(sum_d, ugp, (N+2)*(N+2)*sizeof(double), hipMemcpyHostToDevice);
    //printf("%s\n", cudaGetErrorString(cudaGetLastError()));
    hipDeviceSynchronize();

    tt = omp_get_wtime();
    long Nb = ((N+2)*(N+2)+BLOCK_SIZE-1)/(BLOCK_SIZE);; // each block will compute 1 entry
    std::cout << "Number of blocks: " << Nb << std::endl;
    int gpiter=0;
    double gpres=1.0;
    double sum;
    while (gpres > 1e-6 && gpiter < maxiters){
	gpiter++;
    	gpJacobi<<<Nb,BLOCK_SIZE>>>(up1_d, u_d, f_d, N);
    	hipDeviceSynchronize();
    	// Copy array and compute residual
    	hipMemcpy(u_d,up1_d,(N+2)*(N+2)*sizeof(double),hipMemcpyDeviceToDevice);
    	long Nbr = Nb;
        //Residual calculation
        resvec<<<Nbr,BLOCK_SIZE>>>(sum_d, u_d, f_d, N);
        while (Nbr > 1) {
	   long N1 = Nbr;
	   Nbr = (Nbr+BLOCK_SIZE-1)/(BLOCK_SIZE);
	   reduction_kernel2<<<Nbr,BLOCK_SIZE>>>(sum_d + Nbr, sum_d, N1);
	   sum_d += Nbr;
        }
        hipMemcpyAsync(&sum, sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
	gpres=sqrt(sum)/((double) N);
	if(gpiter%100==0) printf("GPU %10d %10f \n", gpiter, gpres);
    }    
    printf("GPU time = %f s \n", (omp_get_wtime()-tt));
    hipMemcpyAsync(ugp, u_d, (N+2)*(N+2)*sizeof(double), hipMemcpyDeviceToHost);
    double error=0.0;
    //printf("First entry GPU, CPU: %f %f \n", ugp[0], u[0]);
    for (int i=0; i<(N+2)*(N+2); i++){
	error+=(ugp[i]-u[i])*(ugp[i]-u[i]);
	if (i==0) printf("first entry error: %f \n", error);
    }
    printf("Total squared error = %f\n", error);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    hipFree(u_d);
    hipFree(up1_d);
    hipFree(f_d);
    hipHostFree(u);
    hipHostFree(ugp);
    hipHostFree(f);
    return 0;
}
